#include "hip/hip_runtime.h"
%%cuda

#include <iostream>
#include <random>
#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>
#include <hip/hip_runtime.h>
#include <assert.h>

using namespace std;

#define GRID_SIZE 128
#define BLOCK_SIZE 256
#define cudaCheckError() {                                                      \
    hipError_t err = hipGetLastError();                                       \
    if (err != hipSuccess) {                                                   \
        std::cerr << "CUDA error: " << hipGetErrorString(err)                  \
                  << " at " << __FILE__ << ":" << __LINE__ << std::endl;        \
        exit(EXIT_FAILURE);                                                     \
    }                                                                           \
}
#define cudaCheckErrorSync() {                                                  \
    hipDeviceSynchronize();                                                    \
    hipError_t err = hipGetLastError();                                       \
    if (err != hipSuccess) {                                                   \
        std::cerr << "CUDA error: " << hipGetErrorString(err)                  \
                  << " at " << __FILE__ << ":" << __LINE__ << std::endl;        \
        exit(EXIT_FAILURE);                                                     \
    }                                                                           \
}

#define cudaCheckErrorSync() {}
#define cudaCheckError() {}

__global__ void matmul_kernel(const float* A, const float* B, size_t n, size_t m, size_t k, float* output) {
    size_t idi = blockIdx.x * blockDim.x + threadIdx.x;
    size_t idj = blockIdx.y * blockDim.y + threadIdx.y;
    if(idi < n && idj < k) {
        float res = 0;
        for(int idk = 0; idk < m; idk++) {
            res += A[idi * m + idk] * B[idk * k + idj];
        }
        output[idi * k + idj] = res;
    }
}

void matmul(const float* A, const float* B, size_t n, size_t m, size_t k, float* output) {
    float* cu_A;
    float* cu_B;
    float* cu_output;
    size_t A_size = n * m * sizeof(float), B_size = m * k * sizeof(float), out_size = n * k * sizeof(float);
    hipMalloc((void**) &cu_A, A_size);
    cudaCheckError();
    hipMalloc((void**) &cu_B, B_size);
    hipMalloc((void**) &cu_output, out_size);
    cudaCheckError();
    hipMemcpy(cu_A, A, A_size, hipMemcpyHostToDevice);
    hipMemcpy(cu_B, B, B_size, hipMemcpyHostToDevice);
    cudaCheckError();
    hipMemset(cu_output, 0, out_size);
    dim3 grid(GRID_SIZE, GRID_SIZE);
    dim3 block((n + GRID_SIZE - 1) / GRID_SIZE, (k + GRID_SIZE - 1) / GRID_SIZE);
    cerr << (n + GRID_SIZE - 1) / GRID_SIZE << " " << (k + GRID_SIZE - 1) / GRID_SIZE << endl;
    matmul_kernel <<< grid, block >>>(cu_A, cu_B, n, m, k, cu_output);
    cudaCheckErrorSync();

    hipDeviceSynchronize();
    hipMemcpy(output, cu_output, out_size, hipMemcpyDeviceToHost);
    cudaCheckError();
    hipFree(cu_A);
    hipFree(cu_B);
    hipFree(cu_output);
}

const int N = 2000, M = 2000, K = 2000;
const int T = 100;

uniform_real_distribution<float> u(0, 1);
mt19937 rnd(chrono::system_clock::now().time_since_epoch().count());

int main() {
    float* A = new float[N * M];
    float* B = new float[M * K];
    float* C = new float[N * K];
    float* D = new float[N * K];
    for(int i = 0; i < N * M; i++) {
        A[i] = u(rnd);
    }
    for(int i = 0; i < M * K; i++) {
        B[i] = u(rnd);
    }
    cerr << "GENERATE OK!" << endl;

    double st = clock();
    matmul(A, B, N, M, K, C);
    double ed = clock();
    std::cout << (ed - st) / CLOCKS_PER_SEC << std::endl;

    st = clock();
    for(int i = 0; i < N * K; i++) {
        D[i] = 0;
    }
    for(int i = 0; i < N; i++) {
       for(int j = 0; j < M; j++) {
            for(int k = 0; k < K; k++) {
                D[i * K + k] += A[i * M + j] * B[j * K + k];
            }
       }
    }

    ed = clock();
    std::cout << (ed - st) / CLOCKS_PER_SEC << std::endl;

    for(int i = 0; i < N * K; i++) {
        if(fabs(C[i] - D[i]) > 1e-2) {
            std::cout << C[i] << " " << D[i] << " ERROR!" << std::endl;
            break;
        }
    }

    std::cout << "RESULT: " << C[0] << std::endl;
    delete[] A;
    delete[] B;
    delete[] C;
    delete[] D;
    return 0;
}