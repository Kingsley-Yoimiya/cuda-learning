#include "hip/hip_runtime.h"
%%cuda

#include <iostream>
#include <random>
#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <algorithm>

// using namespace std;

#define GRID_SIZE 256
#define BLOCK_SIZE 256
#define BATCH 131072
#define cudaCheckError() {                                                      \
    hipError_t err = hipGetLastError();                                       \
    if (err != hipSuccess) {                                                   \
        std::cerr << "CUDA error: " << hipGetErrorString(err)                  \
                  << " at " << __FILE__ << ":" << __LINE__ << std::endl;        \
        exit(EXIT_FAILURE);                                                     \
    }                                                                           \
}
#define cudaCheckErrorSync() {                                                  \
    hipDeviceSynchronize();                                                    \
    hipError_t err = hipGetLastError();                                       \
    if (err != hipSuccess) {                                                   \
        std::cerr << "CUDA error: " << hipGetErrorString(err)                  \
                  << " at " << __FILE__ << ":" << __LINE__ << std::endl;        \
        exit(EXIT_FAILURE);                                                     \
    }                                                                           \
}

// #define cudaCheckErrorSync() {}
// #define cudaCheckError() {}

int nextPow2(int x) {
    x--;
    for(int i = 1; i < 32; i <<= 1) x |= x >> i;
    return x + 1;
}

__global__ void sort_kernel(const int* input, int* output, int batch, int n, int offset) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stb = id * batch, edb = stb + batch;
    for(int id = stb; id < edb; id++) {
        int st = id * offset * 2, mid = st + offset, ed = mid + offset;
        st = min(st, n), mid = min(mid, n), ed = min(ed, n);
        int pos = st, l = st, r = mid;
        while(l < mid && r < ed) {
            if(input[l] < input[r]) output[pos++] = input[l++];
            else output[pos++] = input[r++];
        }
        while(l < mid) output[pos++] = input[l++];
        while(r < ed)  output[pos++] = input[r++];
    }
}
// 需要优化，直接访问 output 全局数组理论上不快。

void sort(std :: vector < int> &nums) {
    int* cu_in;
    int* cu_out;
    int n = nums.size();
    hipMalloc((void**) &cu_in, n * sizeof(int));
    hipMalloc((void**) &cu_out, n * sizeof(int));
    hipMemcpy(cu_in, nums.data(), n * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(cu_out, 0, n * sizeof(int));
    for(int offset = 1, tot = n + 1 >> 1; tot > 0; offset <<= 1, tot = tot + 1 >> 1) {
        int totcore = min(tot, GRID_SIZE * BLOCK_SIZE);
        int blocknum = min(totcore, BLOCK_SIZE);
        int gridnum = (totcore + blocknum - 1) / blocknum;
        int batch = (tot + blocknum * gridnum - 1) / (blocknum * gridnum);
        // std :: cout << totcore << " " << blocknum << " " << gridnum << " " << batch << std :: endl;
        sort_kernel <<< gridnum, blocknum >>> (cu_in, cu_out, batch, n, offset);
        hipDeviceSynchronize();
        std :: swap(cu_in, cu_out);
        if(tot == 1) break;
    }
    hipMemcpy(nums.data(), cu_in,  n * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(cu_in);
    hipFree(cu_out);
}

const int N = 5e8, M = 2000, K = 2000;
const int T = 100;

std :: uniform_real_distribution<float> u(0, 1);
std :: mt19937 rnd(std :: chrono::system_clock::now().time_since_epoch().count());

int main() {
    std :: vector < int > a, b, c;
    for(int i = 0; i < N; i++) {
        a.emplace_back(rnd() % 1024);
    }
    b = a, c = a;
    std :: cerr << "GENERATE OK!" << a.size() << std :: endl;

    double st = clock();
    sort(a);
    double ed = clock();
    std::cout << (ed - st) / CLOCKS_PER_SEC << std::endl;

    // for(int i = 0; i < 10; i++) cout << a[i] << endl;

    st = clock();
    std :: sort(c.begin(), c.end());
    ed = clock();
    std::cout << (ed - st) / CLOCKS_PER_SEC << std::endl;

    for(int i = 0; i < N; i++) if(a[i] != c[i]) {
        std :: cout << i << " " << a[i] << " " << c[i] << std :: endl;
        break;
    }

    std :: cout << a.size() << " " << c.size() << std :: endl;
    assert(a == c);
    return 0;
}
/*
GENERATE OK!500000000
139.75
280.152
500000000 500000000
*/